#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <cmath>
#include <stdio.h>

__global__ void matrix_vector_product(float* A, float* v1, float* v2, int matrix_size) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (col == 0 && row < matrix_size) {
        float sum = 0;
        for (int i = 0; i < matrix_size; i++) {
            sum += A[row * matrix_size + i] * v1[i];
        }
        v2[row] = sum;
    }
}

int main() {
    float* A, * A_gpu;
    float* v1, * v1_gpu;
    float* v2, * v2_gpu;

    int matrix_size = 40000;

    dim3 block_shape = dim3(32, 32);
    dim3 grid_shape = dim3(std::max(1.0f, std::ceil((float)matrix_size / (float)block_shape.x)),
        std::max(1.0f, std::ceil((float)matrix_size / (float)block_shape.y)));

    // Allocate memory for CPU arrays
    A = new float[matrix_size * matrix_size];
    v1 = new float[matrix_size];
    v2 = new float[matrix_size];

    // Initialize matrices and vectors
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            A[i * matrix_size + j] = (float)i * matrix_size + j;
        }
        v1[i] = (float)i;
    }

    hipMalloc((void**)&A_gpu, matrix_size * matrix_size * sizeof(float));
    hipMalloc((void**)&v1_gpu, matrix_size * sizeof(float));
    hipMalloc((void**)&v2_gpu, matrix_size * sizeof(float));

    // Copy matrices and vectors to GPU
    hipMemcpy(A_gpu, A, matrix_size * matrix_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v1_gpu, v1, matrix_size * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    matrix_vector_product << <grid_shape, block_shape >> > (A_gpu, v1_gpu, v2_gpu, matrix_size);

    // Copy result from GPU to CPU
    hipMemcpy(v2, v2_gpu, matrix_size * sizeof(float), hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < matrix_size; i++) {
        printf("%.2f\n", v2[i]);
    }

    hipFree(A_gpu);
    hipFree(v1_gpu);
    hipFree(v2_gpu);

    delete[] A;
    delete[] v1;
    delete[] v2;

    return 0;
}